#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#if defined(__MACH__) && defined(__APPLE__)
#include <Accelerate/Accelerate.h>
#else
#include <cblas.h>
#endif

#include "datatools.h"		/* helper functions	        */
#include "matadd.h"		/* my matrix add fucntion	*/
#include "multMV.h"		/* my matrix add fucntion	*/

#define NREPEAT 100		/* repeat count for the experiment loop */

#define mytimer clock
#define delta_t(a,b) (1e3 * (b - a) / CLOCKS_PER_SEC)



void matrixVectorTwoGPUS(){
      int    i, m, n, N = NREPEAT;
       //double **A, **B, **C;
      double *h_A, *d0_A, *d1_A; // *B, *C;
      double *h_V1, *d0_V1, *d1_V1,
             *h_V2, *d0_V2, *d1_V2;
       //double tcpu1;

       //clock_t t1, t2;
      int devices;
      hipGetDeviceCount(&devices);

      //for (m = 50; m <= 500; m += 50) {
   	//n = m + 3;
      //p = m;
      m = 4;
      n = 4;
      int A_elems = m*n;
      int V1_elems = n;
      int V2_elems = m;
   	/* Allocate memory */
   	//A = malloc_2d(m, n);
   	//B = malloc_2d(n, p);
   	//C = malloc_2d(m, p);
      //A = malloc(n*m*sizeof(double));
      //B = malloc(n*m*sizeof(double));
      //C = malloc(n*m*sizeof(double));
      //V1 = malloc(n * sizeof(double));
      //V2 = malloc(m * sizeof(double));

      int size_m = n*m*sizeof(double);
      int size_v1 = n*sizeof(double);
      int size_v2 = m*sizeof(double);

      hipHostMalloc((void **)&h_A, size_m);
      hipHostMalloc((void **)&h_V1, size_v1);
      hipHostMalloc((void **)&h_V2, size_v2);
      init_vector(n,h_V1);
      init_vector(m*n,h_A);
      init_vector(m,h_V2);

      hipSetDevice(0);
      hipMalloc((void**)&d0_A, size_m/2);
      hipMalloc((void**)&d0_V1, size_v1);
      hipMalloc((void**)&d0_V2, size_v2/2);
      hipMemcpy(d0_A, h_A, size_m/2, hipMemcpyHostToDevice);
      hipMemcpy(d0_V1, h_V1, size_v1, hipMemcpyHostToDevice);
      hipSetDevice(1);
      hipMalloc((void**)&d1_A, size_m/2);
      hipMalloc((void**)&d1_V1, size_v1);
      hipMalloc((void**)&d1_V2, size_v2/2);
      hipMemcpy(d1_A, h_A + A_elems/2, size_m/2, hipMemcpyHostToDevice);
      hipMemcpy(d1_V1, h_V1, size_v1, hipMemcpyHostToDevice);

      //init_vector(n,h_V1);
      //init_vector(m*n,h_A);
      //init_vector(m,h_V2);

      hipSetDevice(0);

      int blocksize = 1;
      int blocks = ((m / 2) / 32) + 1;
      //multMV<<<blocks,blocksize>>>(m/2,n,d0_A,d0_V1,d0_V2);
      multMV<<<2,1>>>(m/2,n,d0_A,d0_V1,d0_V2);

      hipSetDevice(1);

      //multMV<<<blocks,blocksize>>>(m/2,n,d1_A,d1_V1,d1_V2);
      multMV<<<2,1>>>(m/2,n,d1_A,d1_V1,d1_V2);

      hipSetDevice(0);
      hipMemcpy(h_V2, d0_V2, size_v2/2, hipMemcpyDeviceToHost);
      hipSetDevice(1);
      hipMemcpy(h_V2 + V2_elems/2, d1_V2, size_v2/2, hipMemcpyDeviceToHost);

      hipDeviceSynchronize();

      int k;
      for(k = 0; k < m; k++){
         printf("%f\n", *(h_V2+k));
      }

      hipHostFree(h_A);
      hipHostFree(h_V1);
      hipHostFree(h_V2);
      hipFree(d0_A);
      hipFree(d0_V1);
      hipFree(d0_V2);
      hipFree(d1_A);
      hipFree(d1_V1);
      hipFree(d1_V2);

      /*
   	if (A == NULL || B == NULL | C == NULL) {
   	    fprintf(stderr, "Memory allocation error...\n");
   	    exit(EXIT_FAILURE);
   	}
      */
      /*
      	// initialize with useful data - last argument is reference
      	init_M(m,n,A);

      	//timings for matadd
      	t1 = mytimer();
      	for (i = 0; i < N; i++)
      	    matadd(m, n, A, B, C);
      	t2 = mytimer();
      	tcpu1 = delta_t(t1, t2) / N;

      	check_results("main", m, n, C);

      	// Print n and results
      	printf("%4d %4d %8.3f\n", m, n, tcpu1);
      */
      //init_M(m,n,A);
      //init_M(n,p,B);
      //init_vector(m,V1);

      //multMV(m,n,A,V1,V2);

      //t1 = mytimer();
      //for (i = 0; i < N; i++)
          //multMV(m,n,A,V1,V2);
          //multMM(m,n,p,A,B,C);
      //    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, p, n, 1, A[0], m, B[0], n, 1, C[0], n);
      //t2 = mytimer();
      //tcpu1 = delta_t(t1, t2) / N;
      //printf("%4d %4d %8.3f\n", m, n, tcpu1);
      /*
      for(int k = 0; k < m; k++){
         printf("%f ", **(C+k));
      }
   */
      //hipMemcpy(h_image, d_image, size, hipMemcpyDeviceToHost);


   	/* Free memory */
      /*
   	free_2d(A);
   	free_2d(B);
   	free_2d(C);
      free(V1);
      free(V2);
      */
       //}

       return EXIT_SUCCESS;
}
