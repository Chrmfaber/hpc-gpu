
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void multMV(int m, int n, double *A, double *B, double *C) {
    int i,j;
   i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i >= m) return;
    //for(i = 0; i < m; i++){
      double sum = 0;
      for(j = 0; j < n; j++){
         //printf("%f %f %f \n", A[i][j], B[j], A[i][j]*B[j]);
         sum += A[i*n+j]*B[j];
      }
      C[i] = sum;
      //printf("Hello world! I'm thread %d out of %d in block %d. My global thread id is %d out of %d. C[%d] = %f)\n"
      //      , threadIdx.x, blockDim.x, blockIdx.x, blockIdx.x * blockDim.x + threadIdx.x, blockDim.x*gridDim.x, i, C[i]);
   //}
}



void multMM(int m, int n, int p, double **A, double **B, double **C) {
    int i, j, k;
    double sum;
    //printf("%d %d %d\n", m,n,p);
    for(i = 0; i < m; i++){
      //printf("1");
      for(j = 0; j < p; j++){
         //printf("2");
         sum = 0.0;
         for(k = 0; k < n; k++){
            //printf("3");
            //printf("%f %f %f \n", A[i][k], B[k][j], A[i][k] * B[k][j]);
            sum += A[i][k] * B[k][j];
            //printf("%f\n", sum);
         }
         //printf("%f\n", sum);
         C[i][j] = sum;
      }
   }
}
