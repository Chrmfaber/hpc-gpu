#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#if defined(__MACH__) && defined(__APPLE__)
#include <Accelerate/Accelerate.h>
#else
#include <cblas.h>
#endif

#include "datatools.h"		/* helper functions	        */
#include "matadd.h"		/* my matrix add fucntion	*/
#include "multMV.h"		/* my matrix add fucntion	*/

#define NREPEAT 100		/* repeat count for the experiment loop */

#define mytimer clock
#define delta_t(a,b) (1e3 * (b - a) / CLOCKS_PER_SEC)



int
main(int argc, char *argv[]) {

    int    i, m, n, p, N = NREPEAT;
    //double **A, **B, **C;
    double *h_A, *d_A; // *B, *C;
    double *h_V1, *d_V1, *h_V2, *d_V2;
    double tcpu1;

    clock_t t1, t2;

   //for (m = 50; m <= 500; m += 50) {
	//n = m + 3;
   //p = m;
   m = 3;
   n = 3;

	/* Allocate memory */
	//A = malloc_2d(m, n);
	//B = malloc_2d(n, p);
	//C = malloc_2d(m, p);
   //A = malloc(n*m*sizeof(double));
   //B = malloc(n*m*sizeof(double));
   //C = malloc(n*m*sizeof(double));
   //V1 = malloc(n * sizeof(double));
   //V2 = malloc(m * sizeof(double));

   int size_m = n*m*sizeof(double);
   int size_v1 = n*sizeof(double);
   int size_v2 = m*sizeof(double);

   hipHostMalloc((void **)&h_A, size_m);
   hipHostMalloc((void **)&h_V1, size_v1);
   hipHostMalloc((void **)&h_V2, size_v2);
   hipMalloc((void**)&d_A, size_m);
   hipMalloc((void**)&d_V1, size_v1);
   hipMalloc((void**)&d_V2, size_v2);

   init_vector(n,h_V1);
   init_vector(m*n,h_A);
   init_vector(m,h_V2);

   hipMemcpy(d_A, h_A, size_m, hipMemcpyHostToDevice);
   hipMemcpy(d_V1, h_V1, size_v1, hipMemcpyHostToDevice);

   int blocksize = 32;
   int blocks = (m/32)+1;
   multMV<<<blocks,blocksize>>>(m,n,d_A,d_V1,d_V2);

   hipMemcpy(h_V2, d_V2, size_v2, hipMemcpyDeviceToHost);

   int k;
   for(k = 0; k < m; k++){
      printf("%f\n", *(h_V2+k));
   }

   hipHostFree(h_A);
   hipHostFree(h_V1);
   hipHostFree(h_V2);
   hipFree(d_A);
   hipFree(d_V1);
   hipFree(d_V2);

   /*
	if (A == NULL || B == NULL | C == NULL) {
	    fprintf(stderr, "Memory allocation error...\n");
	    exit(EXIT_FAILURE);
	}
   */
   /*
   	// initialize with useful data - last argument is reference
   	init_M(m,n,A);

   	//timings for matadd
   	t1 = mytimer();
   	for (i = 0; i < N; i++)
   	    matadd(m, n, A, B, C);
   	t2 = mytimer();
   	tcpu1 = delta_t(t1, t2) / N;

   	check_results("main", m, n, C);

   	// Print n and results
   	printf("%4d %4d %8.3f\n", m, n, tcpu1);
   */
   //init_M(m,n,A);
   //init_M(n,p,B);
   //init_vector(m,V1);

   //multMV(m,n,A,V1,V2);

   //t1 = mytimer();
   //for (i = 0; i < N; i++)
       //multMV(m,n,A,V1,V2);
       //multMM(m,n,p,A,B,C);
   //    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, p, n, 1, A[0], m, B[0], n, 1, C[0], n);
   //t2 = mytimer();
   //tcpu1 = delta_t(t1, t2) / N;
   //printf("%4d %4d %8.3f\n", m, n, tcpu1);
   /*
   for(int k = 0; k < m; k++){
      printf("%f ", **(C+k));
   }
*/
   //hipMemcpy(h_image, d_image, size, hipMemcpyDeviceToHost);


	/* Free memory */
   /*
	free_2d(A);
	free_2d(B);
	free_2d(C);
   free(V1);
   free(V2);
   */
    //}

    return EXIT_SUCCESS;
}
