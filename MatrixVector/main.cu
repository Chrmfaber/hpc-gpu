#include <stdio.h>
#include <stdlib.h>
#include "mandel.h"
#include "writepng.h"
#include "mandelgpu.h"



int
main(int argc, char *argv[]) {

    int   width, height;
    int	  max_iter;
    int   *d_image; 
    int   *image;

    int num_blocks = 82;
    int num_threads = 32;

    char s[] = "mandelbrot.png";

    width    = 2601;
    height   = 2601;
    max_iter = 400;
    int size = width * height * sizeof(int);


    // command line argument sets the dimensions of the image
    if ( argc == 2 ) width = height = atoi(argv[1]);

    hipMalloc((void **)&d_image, size);
    hipHostMalloc((void **)&image, size);
    // image = (int *)malloc(size);
    if ( image == NULL ) {
       fprintf(stderr, "memory allocation failed!\n");
       return(1);
    }

    mandelgpu<<<num_blocks, num_threads>>>(width, height, d_image, max_iter);
    hipDeviceSynchronize();
    hipMemcpy(image, d_image, size, hipMemcpyDeviceToHost);

    
    writepng(s, image, width, height);
    hipFree(d_image);
    hipHostFree(image);



    return(0);
}
