extern "C" {

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

void write_result(double *U, int N, double delta, char filename[40]) {
    double u, y, x;
    FILE *matrix=fopen(filename, "w");
    for (int i = 0; i < N; i++) {
        x = -1.0 + i * delta + delta * 0.5;
        for (int j = 0; j < N; j++) {
            y = -1.0 + j * delta + delta * 0.5;
            u = U[i*N + j];
            fprintf(matrix, "%g\t%g\t%g\n", x,y,u);
        }
    }
    fclose(matrix);
}
}

const int device0 = 0;

void __global__ jacobi_gpu1(int N, double delta, int kMAX, double *f, double *u_new, double *u_old) {
    int j,i;
    double scalar = 1.0 / 4;
    for (i = 1; i <= N; i++) {
        for (j = 1; j <= N; j++) {
            // Update u
            u_new[i*N + j] = scalar * (u_old[(i-1)*N + j] + u_old[(i+1)*N + j] + u_old[i*N + (j-1)] + u_old[i*N + (j+1)] + delta*delta*f[i*N + j]);
        }
    }
}


int main(int argc, char *argv[]) {

    // warm up:
    double *dummy_d;
    hipSetDevice(device0);
    hipMalloc((void**)&dummy_d, 0);

    int i, j, N, kMAX;

    if (argc == 3) {
        N = atoi(argv[1]);
        kMAX = atoi(argv[2]);
    }
    else {
        // use default N
        N = 200;
        kMAX = 5000;
    }
    double delta = (2.0 / N) * (2.0 / N);

    // allocate mem
    double *h_f, *h_u_new, *h_u_old, *d_f, *d_u_new, *d_u_old;

    int size_f = N * N * sizeof(double);
    int size_u_new = N * N * sizeof(double);
    int size_u_old = N * N * sizeof(double);

    //Allocate memory on device
    hipSetDevice(device0);
    hipMalloc((void**)&d_f, size_f);
    hipMalloc((void**)&d_u_new, size_u_new);
    hipMalloc((void**)&d_u_old, size_u_old);
    //Allocate memory on host
    hipHostMalloc((void**)&h_f, size_f, hipHostMallocDefault);
    hipHostMalloc((void**)&h_u_new, size_u_new, hipHostMallocDefault);
    hipHostMalloc((void**)&h_u_old, size_u_old, hipHostMallocDefault);

    // initilize boarder
    for (i = 0; i < N; i++){
        for (j = 0; j < N; j++){
            if (i >= N * 0.5  &&  i <= N * 2.0/3.0  &&  j >= N * 1.0/6.0  &&  j <= N * 1.0/3.0)
                h_f[i*N + j] = 200.0;
            else
                h_f[i*N + j] = 0.0;

            if (i == (N - 1) || i == 0 || j == (N - 1)){
                h_u_new[i*N + j] = 20.0;
                h_u_old[i*N + j] = 20.0;
            }
            else{
                h_u_new[i*N + j] = 0.0;
                h_u_old[i*N + j] = 0.0;
            }
        }
    }


    //Copy memory CPU -> GPU
    double time_tmp = omp_get_wtime();
    hipMemcpy(d_f, h_f, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_u_new, h_u_new, size_u_old, hipMemcpyHostToDevice);
    hipMemcpy(d_u_old, h_u_old, size_u_old, hipMemcpyHostToDevice);
    double time_IO_1 = omp_get_wtime() - time_tmp;

    // do program
    int k = 0;
    double *temp, time_compute = omp_get_wtime();
    while (k < kMAX) {
        // Set u_old = u
        temp = d_u_new;
        d_u_new = d_u_old;
        d_u_old = temp;
        jacobi_gpu1<<<1,1>>>(N, delta, kMAX, d_f, d_u_new, d_u_old);
        hipDeviceSynchronize();
        k++;
    }/* end while */
    double tot_time_compute = omp_get_wtime() - time_compute;
    // end program

    //Copy memory GPU -> CPU
    time_tmp = omp_get_wtime();
    hipMemcpy(h_u_new, d_u_new, size_u_new, hipMemcpyDeviceToHost);
    double time_IO_2 = omp_get_wtime() - time_tmp;

    tot_time_compute += time_IO_1 + time_IO_2;

    // stats
    double GB = 1.0e-09;
    double flop = kMAX * (double)(N) * (double)(N) * 10.0;
    double gflops  = (flop / tot_time_compute) * GB;
    double memory  = size_f + size_u_new + size_u_old;
    double memoryGBs  = memory * GB * (1 / tot_time_compute);

    printf("%g\t", N);
    printf("%g\t", memory); // footprint
    printf("%g\t", gflops); // Gflops
    //printf("%g\t", memoryGBs); // bandwidth GB/s
    printf("%g\t", tot_time_compute); // total time
    //printf("%g\t", time_IO_1 + time_IO_2); // I/O time
    printf("%g\t", tot_time_compute); // compute time
    printf("# GPU1\n");


    //write_result(h_u_new, N, delta, "jacobi_gpu1.dat");

    // free mem
    hipFree(d_f), hipFree(d_u_new), hipFree(d_u_old);
    hipHostFree(h_f), hipHostFree(h_u_new), hipHostFree(h_u_old);
    // end program
    return(0);
}
