#include "hip/hip_runtime.h"
extern "C" {
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>


void write_matrix(double *U, int N, char filename[40]) {
<<<<<<< HEAD
  double u;
  double delta = (2.0 / N);
  FILE *matrix = fopen(filename, "w");
  for (int i = 0; i < N; i++) {
    fprintf(matrix, "\n");
    for (int j = 0; j < N; j++) {
      u = U[i * N + j];
      fprintf(matrix, "%6.2f\t", u);
=======
    double u;
    FILE *matrix=fopen(filename, "w");
    for (int i = 0; i < N; i++) {
        fprintf(matrix, "\n");
        for (int j = 0; j < N; j++) {
            u = U[i*N + j];
            fprintf(matrix, "%6.2f\t",u);
        }
>>>>>>> c35594f8edc18d82038f020126488e7de339070c
    }
  }
  fclose(matrix);
}
}

const int device0 = 0;
const int device1 = 1;
#define BLOCK_SIZE 16

<<<<<<< HEAD
void __global__ jacobi_gpu3_d0(int N, double delta, int kMAX, double *f,
                               double *u_new, double *u_old, double *d1_u_old) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i <= (N * 0.5 - 1) && j <= (N - 1) && i > 0 && j > 0) {
    u_new[i * N + j] = 0.25 * (u_old[(i - 1) * N + j] + u_old[(i + 1) * N + j] +
                               u_old[i * N + (j - 1)] + u_old[i * N + (j + 1)] +
                               delta * f[i * N + j]);
  } else if (i == (N * 0.5 - 1) && j < (N - 1) && j > 0) {
    u_new[i * N + j] =
        0.25 * (u_old[(i - 1) * N + j] + d1_u_old[j] + u_old[i * N + (j - 1)] +
                u_old[i * N + (j + 1)] + delta * f[i * N + j]);
  }
=======

void __global__ jacobi_gpu3_d0(int N, double delta, int kMAX, double *f, double *u_new, double *u_old, double *d1_u_old) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i <(N*0.5-1) && j < (N-1) && i > 0 && j > 0) {
        u_new[i*N + j] = 0.25 * (u_old[(i-1)*N + j] + u_old[(i+1)*N + j] + u_old[i*N + (j-1)] + u_old[i*N + (j+1)] + delta*f[i*N + j]);
    }
    else if (i == (N/2-1) && j < (N-1) && j > 0) {

        u_new[i*N + j] = 0.25 * (u_old[(i-1)*N + j] + d1_u_old[j] + u_old[i*N + (j-1)] + u_old[i*N + (j+1)] + delta*f[i*N + j]);
    }
>>>>>>> c35594f8edc18d82038f020126488e7de339070c
}

void __global__ jacobi_gpu3_d1(int N, double delta, int kMAX, double *f,
                               double *u_new, double *u_old, double *d0_u_old) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < (N * 0.5 - 1) && j < (N - 1) && i > 0 && j > 0) { // i < N/2
    u_new[i * N + j] = 0.25 * (u_old[(i - 1) * N + j] + u_old[(i + 1) * N + j] +
                               u_old[i * N + (j - 1)] + u_old[i * N + (j + 1)] +
                               delta * f[i * N + j]);
  } else if (i == 0 && j < (N - 1) && j > 0) {
    u_new[i * N + j] = 0.25 * (d0_u_old[(N / 2 - 1) * N + j] +
                               u_old[(i + 1) * N + j] + u_old[i * N + (j - 1)] +
                               u_old[i * N + (j + 1)] + delta * f[i * N + j]);
  }
}

int main(int argc, char *argv[]) {

<<<<<<< HEAD
  // warm up:
  double *dummy_d;
  hipSetDevice(device0);
  hipMalloc((void **)&dummy_d, 0);
  hipSetDevice(device1);
  hipMalloc((void **)&dummy_d, 0);

  int kMAX, N, i, j;

  if (argc == 3) {
    N = atoi(argv[1]) + 2;
    kMAX = atoi(argv[2]);
  } else {
    // use default N
    N = 200;
    kMAX = 5000;
  }
  double delta = (2.0 / N) * (2.0 / N);

  // allocate mem
  double *h_f, *h_u_new, *h_u_old;
  double *d0_f, *d0_u_new, *d0_u_old, *d1_f, *d1_u_new, *d1_u_old;

  int size_f = N * N * sizeof(double);
  int size_u_new = N * N * sizeof(double);
  int size_u_old = N * N * sizeof(double);
  int size_f_p2 = N * N * 0.5;
  int size_u_new_p2 = N * N * 0.5;
  int size_u_old_p2 = N * N * 0.5;

  // Allocate memory on device
  hipSetDevice(device0);
  hipMalloc((void **)&d0_f, size_f / 2);
  hipMalloc((void **)&d0_u_new, size_u_new / 2);
  hipMalloc((void **)&d0_u_old, size_u_old / 2);
  hipSetDevice(device1);
  hipMalloc((void **)&d1_f, size_f / 2);
  hipMalloc((void **)&d1_u_new, size_u_new / 2);
  hipMalloc((void **)&d1_u_old, size_u_old / 2);
  // Allocate memory on host
  hipHostMalloc((void **)&h_f, size_f);
  hipHostMalloc((void **)&h_u_new, size_u_new);
  hipHostMalloc((void **)&h_u_old, size_u_old);

  // initialize boarder
  for (i = 0; i < N; i++) {
    for (j = 0; j < N; j++) {
      if (i >= N * 0.5 && i <= N * 2.0 / 3.0 && j >= N * 1.0 / 6.0 &&
          j <= N * 1.0 / 3.0)
        h_f[i * N + j] = 200.0;
      else
        h_f[i * N + j] = 0.0;

      if (i == (N - 1) || i == 0 || j == (N - 1)) {
        h_u_new[i * N + j] = 20.0;
        h_u_old[i * N + j] = 20.0;
      } else {
        h_u_new[i * N + j] = 0.0;
        h_u_old[i * N + j] = 0.0;
      }
=======
    // warm up:
    double *dummy_d;
    hipSetDevice(device0);
    hipMalloc((void**)&dummy_d, 0);
    hipSetDevice(device1);
    hipMalloc((void**)&dummy_d, 0);

    int kMAX, N,i,j;

    N = atoi(argv[1])+2;
    kMAX = atoi(argv[2]);

    double delta = (2.0 / N) * (2.0 / N);

    // allocate mem
    double *h_f, *h_u_new, *h_u_old;
    double *d0_f, *d0_u_new, *d0_u_old, *d1_f, *d1_u_new, *d1_u_old;

    int size = N * N * sizeof(double);
    int size_p2 = N*N*0.5;


    //Allocate memory on device
    hipSetDevice(device0);
    hipMalloc((void**)&d0_f, size/2);
    hipMalloc((void**)&d0_u_new, size/2);
    hipMalloc((void**)&d0_u_old, size/2);
    hipSetDevice(device1);
    hipMalloc((void**)&d1_f, size/2);
    hipMalloc((void**)&d1_u_new, size/2);
    hipMalloc((void**)&d1_u_old, size/2);
    //Allocate memory on host
    hipHostMalloc((void**)&h_f, size);
    hipHostMalloc((void**)&h_u_new, size);
    hipHostMalloc((void**)&h_u_old, size);

    // initialize boarder
    for (i = 0; i < N; i++){
        for (j = 0; j < N; j++){
            if (i >= N * 0.5  &&  i <= N * 2.0/3.0  &&  j >= N * 1.0/6.0  &&  j <= N * 1.0/3.0)
                h_f[i*N + j] = 200.0;
            else
                h_f[i*N + j] = 0.0;

            if (i == (N - 1) || i == 0 || j == (N - 1)){
                h_u_new[i*N + j] = 20.0;
                h_u_old[i*N + j] = 20.0;
            }
            else{
                h_u_new[i*N + j] = 0.0;
                h_u_old[i*N + j] = 0.0;
            }
        }
>>>>>>> c35594f8edc18d82038f020126488e7de339070c
    }
  }

  // Copy memory host -> device
  double time_tmp = omp_get_wtime();
  hipSetDevice(device0);
  hipMemcpy(d0_f, h_f, size_f / 2, hipMemcpyHostToDevice);
  hipMemcpy(d0_u_new, h_u_new, size_u_new / 2, hipMemcpyHostToDevice);
  hipMemcpy(d0_u_old, h_u_old, size_u_old / 2, hipMemcpyHostToDevice);

  hipSetDevice(device1);
  hipMemcpy(d1_f, h_f + size_f_p2, size_f / 2, hipMemcpyHostToDevice);
  hipMemcpy(d1_u_new, h_u_new + size_u_new_p2, size_u_new / 2,
             hipMemcpyHostToDevice);
  hipMemcpy(d1_u_old, h_u_old + size_u_old_p2, size_u_old / 2,
             hipMemcpyHostToDevice);
  double time_IO_1 = omp_get_wtime() - time_tmp;

  // peer enable
  hipSetDevice(device0);
  hipDeviceEnablePeerAccess(device1, 0);
  hipSetDevice(device1);
  hipDeviceEnablePeerAccess(device0, 0);

  // do program
  int k = 0;
  dim3 dim_grid(((N + BLOCK_SIZE - 1) / BLOCK_SIZE),
                ((N / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE));
  dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
  double *temp_p;
  double time_compute = omp_get_wtime();
  while (k < kMAX) {
    // Set u_old = u device 0
    temp_p = d0_u_new;
    d0_u_new = d0_u_old;
    d0_u_old = temp_p;
    // Set u_old = u device 0
    temp_p = d1_u_new;
    d1_u_new = d1_u_old;
    d1_u_old = temp_p;

<<<<<<< HEAD
    hipSetDevice(device0);
    jacobi_gpu3_d0<<<dim_grid, dim_block>>>(N, delta, kMAX, d0_f, d0_u_new,
                                            d0_u_old, d1_u_old);
    hipSetDevice(device1);
    jacobi_gpu3_d1<<<dim_grid, dim_block>>>(N, delta, kMAX, d1_f, d1_u_new,
                                            d1_u_old, d0_u_old);
    hipDeviceSynchronize();
    hipSetDevice(device0);
    hipDeviceSynchronize();
    k++;
  } /* end while */
  double tot_time_compute = omp_get_wtime() - time_compute;
  // end program

  // Copy memory host -> device
  time_tmp = omp_get_wtime();
  hipSetDevice(device0);
  hipMemcpy(h_u_new, d0_u_new, size_u_new / 2, hipMemcpyDeviceToHost);
  hipSetDevice(device1);
  hipMemcpy(h_u_new + size_u_new_p2, d1_u_new, size_u_new / 2,
             hipMemcpyDeviceToHost);
  double time_IO_2 = omp_get_wtime() - time_tmp;

  tot_time_compute += time_IO_1 + time_IO_2;

  // stats
  double GB = 1.0e-09;
  double flop = kMAX * (double)(N) * (double)(N)*10.0;
  double gflops = (flop / tot_time_compute) * GB;
  double memory = size_f + size_u_new + size_u_old;
  double memoryGBs = memory * GB * (1 / tot_time_compute);

  printf("%d\t", N);
  printf("%g\t", memory); // footprint
  printf("%g\t", gflops); // Gflops
  // printf("%g\t", memoryGBs); // bandwidth GB/s
  printf("%g\t", tot_time_compute); // total time
  // printf("%g\t", time_IO_1 + time_IO_2); // I/O time
  // printf("%g\t", tot_time_compute); // compute time
  printf("# GPU3\n");
  write_matrix(h_u_new, N, "gpu3.dat");

  // peer enable
  hipSetDevice(device0);
  hipDeviceDisablePeerAccess(device1);
  hipSetDevice(device1);
  hipDeviceDisablePeerAccess(device0);

  // free mem
  hipFree(d0_f), hipFree(d0_u_new), hipFree(d0_u_old);
  hipFree(d1_f), hipFree(d1_u_new), hipFree(d1_u_old);
  hipHostFree(h_f), hipHostFree(h_u_new), hipHostFree(h_u_old);
  // end program
  return (0);
=======
    //Copy memory CPU -> GPU
    double time_tmp = omp_get_wtime();
    hipSetDevice(device0);
    hipMemcpy(d0_f, h_f, size/2, hipMemcpyHostToDevice);
    hipMemcpy(d0_u_new, h_u_new, size/2, hipMemcpyHostToDevice);
    hipMemcpy(d0_u_old, h_u_old, size/2, hipMemcpyHostToDevice);
    hipSetDevice(device1);
    hipMemcpy(d1_f, h_f + size_p2, size/2, hipMemcpyHostToDevice);
    hipMemcpy(d1_u_new, h_u_new + size_p2, size/2, hipMemcpyHostToDevice);
    hipMemcpy(d1_u_old, h_u_old + size_p2, size/2, hipMemcpyHostToDevice);
    double time_IO_1 = omp_get_wtime() - time_tmp;

    // peer enable
    hipSetDevice(device0);
    hipDeviceEnablePeerAccess(device1,0);
    hipSetDevice(device1);
    hipDeviceEnablePeerAccess(device0,0);

    // do program
    int k = 0;
    dim3 dim_grid(((N +BLOCK_SIZE-1) / BLOCK_SIZE), ((N/2+BLOCK_SIZE-1) / BLOCK_SIZE));
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
    double *temp_p;
    double time_compute = omp_get_wtime();
    while (k < kMAX) {
        // Set u_old = u device 0
        temp_p = d0_u_new;
        d0_u_new = d0_u_old;
        d0_u_old = temp_p;
        // Set u_old = u device 0
        temp_p = d1_u_new;
        d1_u_new = d1_u_old;
        d1_u_old = temp_p;

        hipSetDevice(device0);
        jacobi_gpu3_d0<<<dim_grid, dim_block>>>(N, delta, kMAX, d0_f, d0_u_new, d0_u_old, d1_u_old);
        hipSetDevice(device1);
        jacobi_gpu3_d1<<<dim_grid, dim_block>>>(N, delta, kMAX, d1_f, d1_u_new, d1_u_old, d0_u_old);
        hipDeviceSynchronize();
        hipSetDevice(device0);
        hipDeviceSynchronize();
        k++;
    }/* end while */
    double tot_time_compute = omp_get_wtime() - time_compute;
    // end program

    //Copy memory GPU -> CPU
    time_tmp = omp_get_wtime();
    hipSetDevice(device0);
    hipMemcpy(h_u_new, d0_u_new, size/2, hipMemcpyDeviceToHost);
    hipSetDevice(device1);
    hipMemcpy(h_u_new + size_p2, d1_u_new, size/2, hipMemcpyDeviceToHost);
    double time_IO_2 = omp_get_wtime() - time_tmp;

    tot_time_compute += time_IO_1 + time_IO_2;

    // stats
    double GB = 1.0e-09;
    double flop = kMAX * (double)(N) * (double)(N) * 10.0;
    double gflops  = (flop / tot_time_compute) * GB;
    double memory  = size*3;
    double memoryGBs  = memory * GB * (1 / tot_time_compute);

    printf("%d\t", N);
    printf("%g\t", memory); // footprint
    printf("%g\t", gflops); // Gflops
    //printf("%g\t", memoryGBs); // bandwidth GB/s
    printf("%g\t", tot_time_compute); // total time
    //printf("%g\t", time_IO_1 + time_IO_2); // I/O time
    //printf("%g\t", tot_time_compute); // compute time
    printf("# GPU3\n");

    //To validate result we can write out the matrix
    write_matrix(h_u_new, N, "gpu3.dat");

    // peer enable
    hipSetDevice(device0);
    hipDeviceDisablePeerAccess(device1);
    hipSetDevice(device1);
    hipDeviceDisablePeerAccess(device0);

    // free mem
    hipFree(d0_f), hipFree(d0_u_new), hipFree(d0_u_old);
    hipFree(d1_f), hipFree(d1_u_new), hipFree(d1_u_old);
    hipHostFree(h_f), hipHostFree(h_u_new), hipHostFree(h_u_old);
    // end program
    return(0);
>>>>>>> c35594f8edc18d82038f020126488e7de339070c
}
