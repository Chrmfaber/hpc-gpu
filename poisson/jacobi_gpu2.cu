#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
  void write_matrix(double *U, int N, char filename[40]) {
      double u;
      FILE *matrix=fopen(filename, "w");
      for (int i = 0; i < N; i++) {
          fprintf(matrix, "\n");
          for (int j = 0; j < N; j++) {
              u = U[i*N + j];
              fprintf(matrix, "%6.2f\t",u);
          }
      }
      fclose(matrix);
  }
}

const int device0 = 0;
#define BLOCK_SIZE 16


void __global__ jacobi_gpu2(int N, double delta, int kMAX, double *f, double *u_new, double *u_old) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    double scalar = 1.0/4;
    if (i < N-1 && j < N-1 && i > 0 && j > 0) {
        u_new[i*N + j] = scalar * (u_old[(i-1)*N + j] + u_old[(i+1)*N + j] + u_old[i*N + (j-1)] + u_old[i*N + (j+1)] +delta*f[i*N + j]);
    }
}


int main(int argc, char *argv[]) {

    // warm up:
    double *dummy_d;
    hipSetDevice(device0);
    hipMalloc((void**)&dummy_d, 0);

    int kMAX, N,i,j;

    N = atoi(argv[1]) +2 ;
    kMAX = atoi(argv[2]);


    double delta = (2.0 / N) * (2.0 / N);

    // allocate mem
    double *h_f, *h_u_new, *h_u_old, *d_f, *d_u_new, *d_u_old;

    int size = N * N * sizeof(double);

    //Allocate memory on device
    hipSetDevice(device0);
    hipMalloc((void**)&d_f, size);
    hipMalloc((void**)&d_u_new, size);
    hipMalloc((void**)&d_u_old, size);
    //Allocate memory on host
    hipHostMalloc((void**)&h_f, size);
    hipHostMalloc((void**)&h_u_new, size);
    hipHostMalloc((void**)&h_u_old, size);

    // initilize boarder
    for (i = 0; i <N; i++){
        for (j = 0; j < N; j++){
            if (i >= N * 0.5  &&  i <= N * 2.0/3.0  &&  j >= N * 1.0/6.0  &&  j <= N * 1.0/3.0)
                h_f[i*N + j] = 200.0;
            else
                h_f[i*N + j] = 0.0;

            if (i == (N - 1) || i == 0 || j == (N - 1)){
                h_u_new[i*N + j] = 20.0;
                h_u_old[i*N + j] = 20.0;
            }
            else{
                h_u_new[i*N + j] = 0.0;
                h_u_old[i*N + j] = 0.0;
            }
        }
    }

    //Copy memory CPU -> GPU
    double time_tmp = omp_get_wtime();
    hipMemcpy(d_f, h_f, size, hipMemcpyHostToDevice);
    hipMemcpy(d_u_new, h_u_new, size hipMemcpyHostToDevice);
    hipMemcpy(d_u_old, h_u_old, size, hipMemcpyHostToDevice);
    double time_IO_1 = omp_get_wtime() - time_tmp;

    // do program
    int k = 0;
    dim3 dim_grid(((N+BLOCK_SIZE-1) / BLOCK_SIZE), ((N+BLOCK_SIZE-1) / BLOCK_SIZE));
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
    double *temp, time_compute = omp_get_wtime();
    while (k < kMAX) {
        // Set u_old = u
        temp = d_u_new;
        d_u_new = d_u_old;
        d_u_old = temp;
        jacobi_gpu2<<<dim_grid,dim_block>>>(N, delta, kMAX, d_f, d_u_new, d_u_old);
        hipDeviceSynchronize();
        k++;
    }/* end while */
    double tot_time_compute = omp_get_wtime() - time_compute;
    // end program

    //Copy memory GPU -> CPU
    time_tmp = omp_get_wtime();
    hipMemcpy(h_u_new, d_u_new, size, hipMemcpyDeviceToHost);
    double time_IO_2 = omp_get_wtime() - time_tmp;

    tot_time_compute += time_IO_1 + time_IO_2;

    // stats
    double GB = 1.0e-09;
    double flop = kMAX * (double)(N-2) * (double)(N-2) * 10.0;
    double gflops  = (flop / tot_time_compute) * GB;
    double memory  = size*3;
    double memoryGBs  = memory * GB * (1 / tot_time_compute);

    printf("%d\t", N);
    printf("%g\t", memory); // footprint
    printf("%g\t", gflops); // Gflops
    //printf("%g\t", memoryGBs); // bandwidth GB/s
    printf("%g\t", tot_time_compute); // total time
    //printf("%g\t", time_IO_1 + time_IO_2); // I/O time
    //printf("%g\t", tot_time_compute); // compute time
    printf("# GPU2\n");

    write_matrix(h_u_new, N, "gpu2.dat");

    // free allocated mem
    hipFree(d_f), hipFree(d_u_new), hipFree(d_u_old);
    hipHostFree(h_f), hipHostFree(h_u_new), hipHostFree(h_u_old);
    // end program
    return(0);
}
