#include <hipblas.h>
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <cstdlib>
#include <cstdio>
#include <datatools.h>

void matmult_gpulib(const float *A, const float *B, float *C, const int m, const int k, const int n) {
      int lda=m,ldb=k,ldc=m;
      const float alf = 1.0;
      const float bet = 0.0;
      const float *alpha = &alf;
      const float *beta = &bet;

     // Create a handle for CUBLAS
     hipblasHandle_t handle;
     hipblasCreate(&handle);

     // Do the actual multiplication
     hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

     // Destroy the handle
     hipblasDestroy(handle);

   }
