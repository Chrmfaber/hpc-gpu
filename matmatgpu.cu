
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matmatgpu(int n, int m, double *A, double *B, double *C) {
  int i, j;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= m)
    return;

  double sum = 0;
  for (j = 0; j < n; j++) {
    sum += A[i * n + j] * B[i * n + j];
  }
  C[i] = sum;
};
