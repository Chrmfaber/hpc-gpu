
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matmatgpu(int n, int m, double *A, double *B, double *C) {
  int i, j;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= m)
    return;

  double sum = 0;
  for (j = 0; j < n; j++) {
    printf("A[i * n + j] = %f, B[i * n + j] = %f \n",A[i * n + j], B[i * n + j]);
    sum += A[i * n + j] * B[i * n + j];
  }
  C[i] = sum;
};
