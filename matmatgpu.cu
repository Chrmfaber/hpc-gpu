
#include <hip/hip_runtime.h>

__global__ void matmatgpu(int n, int m, double *A, double *B, double *C) {
  int i, j;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= m)
    return;

  double sum = 0;
  for (j = 0; j < n; j++) {
    sum += A[i * n + j] * B[i * n + j];
  }
  C[i] = sum;
  // printf("Hello world! I'm thread %d out of %d in block %d. My global thread
  // id is %d out of %d. C[%d] = %f)\n"
  //      , threadIdx.x, blockDim.x, blockIdx.x, blockIdx.x * blockDim.x +
  //      threadIdx.x, blockDim.x*gridDim.x, i, C[i]);
  //}
};
