#include <stdio.h>
#include <stdlib.h>
#include "matmatgpu.h"



int
main(int argc, char *argv[]) {

    int n = 32;
    int m = 32;

    int num_blocks = 82;
    int num_threads = 32;

    double **A, **B, **C;

    int size = n * m * sizeof(int);


    // command line argument sets the dimensions of the image
    if ( argc == 2 ) m = n = atoi(argv[1]);

    hipMalloc((void **)&d_image, size);
    hipHostMalloc((void **)&image, size);
    // image = (int *)malloc(size);
    if ( image == NULL ) {
       fprintf(stderr, "memory allocation failed!\n");
       return(1);
    }

    mandelgpu<<<num_blocks, num_threads>>>(width, height, d_image, max_iter);
    hipDeviceSynchronize();
    hipMemcpy(image, d_image, size, hipMemcpyDeviceToHost);


    writepng(s, image, width, height);
    hipFree(d_image);
    hipHostFree(image);



    return(0);
}
