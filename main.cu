#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "datatools.h"
#include "matmatgpu.h"

#define mytimer clock
#define delta_t(a, b) (1e3 * (b - a) / CLOCKS_PER_SEC)

int main(int argc, char *argv[]) {

  int n = 10;
  int m = 10;

  double tcpu1;

  clock_t t1, t2;

  // command line argument sets the dimensions of the image
  if (argc == 2)
    m = n = atoi(argv[1]);

  // Kernel config
  int num_blocks = 82;
  int num_threads = 32;

  // Single indexed matrices (host)
  double *A, *B, *C;

  // device
  double *d_A, *d_B, *d_C;

  // Size of matrix
  int size = n * m * sizeof(double);

  // Memory allocation (host)
  hipHostMalloc((void **)&A, size);
  hipHostMalloc((void **)&B, size);
  hipHostMalloc((void **)&C, size);

  // device
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  hipMalloc((void **)&d_C, size);

  // Initalize A with 1s and B with 2s
  init_data(m, n, A, B);

  print_matrix(m, n, A);
  print_matrix(m, n, B);

  // Copy to device
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  if (A == NULL || B == NULL || C == NULL) {
    fprintf(stderr, "memory allocation failed!\n");
    return (1);
  }

  matmatgpu<<<num_blocks, num_threads>>>(n, m, d_A, d_B, d_C);
  hipDeviceSynchronize();

  // Copy back to host
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  print_matrix(m, n, C);

  // Free memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);

  return (0);
}
