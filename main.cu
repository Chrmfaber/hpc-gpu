#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>

#include "matmatgpu.h"

#define mytimer clock
#define delta_t(a,b) (1e3 * (b - a) / CLOCKS_PER_SEC)


int
main(int argc, char *argv[]) {

    int n = 32;
    int m = 32;

    double tcpu1;

    clock_t t1, t2;

    // command line argument sets the dimensions of the image
    if ( argc == 2 ) m = n = atoi(argv[1]);

    // Kernel config
    int num_blocks = 82;
    int num_threads = 32;

    // Single indexed matrices (host)
    double *A, *B, *C;

    // device
    double *d_A, *d_B, *d_C;

    // Size of matrix
    int size = n * m * sizeof(double);

    // Memory allocation (host)
    hipHostMalloc((void **)&A, size);
    hipHostMalloc((void **)&B, size);
    hipHostMalloc((void **)&C, size);

    // device
    hipMalloc((void **)&d_A, size)
    hipMalloc((void **)&d_B, size)
    hipMalloc((void **)&d_C, size)

    // Initalize A with 1s and B with 2s
    init_data(m, n, A, B);

    // Copy to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice)
    hipMemcpy(d_A, B, size, hipMemcpyHostToDevice)

    if ( A == NULL  || B == NULL || C == NULL) {
       fprintf(stderr, "memory allocation failed!\n");
       return(1);
    }

    mandelgpu<<<num_blocks, num_threads>>>(width, height, d_image, max_iter);
    hipDeviceSynchronize();

    // Copy back to host
    hipMemcpy(C, d_C , size, hipMemcpyDeviceToHost)

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);




    return(0);
}
