
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void d_gpu1(int m, int n, int p, double *A, double *B, double *C) {
  int i, j, k;

  double sum = 0;
  for (i = 0; i < m; i++) {
    for (j = 0; j < n; j++) {

      for (k = 0; k < p; k++) {
        sum += A[i * p + k] * B[k * m + j];

      }
      C[i * n + j] = sum;
    }
  }
};

extern "C" {
void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C) {
  // Copy to device logic

  // Kernel config
  // TODO: set dynamically
  int num_blocks = 1;
  int num_threads = 1;

  // Size of matrix
  int size_A = m * k * sizeof(double);
  int size_B = k * n * sizeof(double);
  int size_C = m * n * sizeof(double);

  // device matrices initialized
  double *d_A, *d_B, *d_C;

  // device matrices memory allocation
  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_B, size_B);
  hipMalloc((void **)&d_C, size_C);

  // Copy to device
  hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_A, B, size_B, hipMemcpyHostToDevice);

  // launch kernel
  d_gpu1<<<num_blocks, num_threads>>>(m, n, k, d_A, d_B, d_C);

  // sync threads
  hipDeviceSynchronize();

  // Copy back to host
  hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

  // Free memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
};
}
