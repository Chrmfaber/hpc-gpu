#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void d_gpu1(int m, int n, int k, double *A, double *B, double *C) {
  int i, j, l;

  if (i >= m)
    return;

  double sum = 0;
  for (i = 0, i < m; i++) {
    for (j = 0; j < n; j++) {
      for (l = 0; l < k; l++) {
        sum += A[i * m + k] * B[l * n + j];
      }
      C[i * m + j] = sum;
    }
  }
};

void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C) {
  // Copy to device logic

  // Kernel config
  // TODO: set dynamically
  int num_blocks = 1;
  int num_threads = 1;

  // Size of matrix
  int size_A = m * k * sizeof(double);
  int size_B = k * n * sizeof(double);
  int size_C = m * n * sizeof(double);

  // device matrices initialized
  double *d_A, *d_B, *d_C;

  // device matrices memory allocation
  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_B, size_B);
  hipMalloc((void **)&d_C, size_C);

  // Copy to device
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_A, B, size, hipMemcpyHostToDevice);

  // launch kernel
  d_gpu1<<<num_blocks, num_threads>>>(m, n, k, A, B, C);

  // sync threads
  hipDeviceSynchronize();

  // Copy back to host
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  // Free memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Necessary?
  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);

  return (0);
};
