
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>

__global__ void d_gpu1(int m, int n, int k, double *A, double *B, double *C) {
  int i, j, l;

  for (i = 0; i < m; i++) {
    for (j = 0; j < n; j++) {
      double sum = 0;
      for (l = 0; l < k; l++) {
        sum += A[i * k + l] * B[l * n + j];
      }
      C[i * n + j] = sum;
    }
  }
};

extern "C" {
void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C) {
  // Copy to device logic
  int i, j;




  // Kernel config
  // TODO: set dynamically
  int num_blocks = 1;
  int num_threads = 1;

  // Size of matrix
  int size_A = m * k * sizeof(double);
  int size_B = k * n * sizeof(double);
  int size_C = m * n * sizeof(double);

  // device matrices initialized
  double *d_A, *d_B, *d_C;

  // device matrices memory allocation
  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_B, size_B);
  hipMalloc((void **)&d_C, size_C);

  // Copy to device
  hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

  double time_start_gpu1 = omp_get_wtime();
  // launch kernel
  d_gpu1<<<num_blocks, num_threads>>>(m, n, k, d_A, d_B, d_C);

  // sync threads
  hipDeviceSynchronize();

  double gpu1_time = omp_get_wtime()-time_start_gpu1;

  // Copy back to host
  hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

  printf("GPUTime = %f\n", gpu1_time);

  // Free memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
};
}
