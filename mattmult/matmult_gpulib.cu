
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <cstdlib>
#include <cstdio>

extern "C" { void matmult_gpulib(const double *A, const double *B, double *C, const int m, const int k, const int n) {
      int lda=m,ldb=k,ldc=m;
      const double alf = 1.0;
      const double bet = 0.0;
      const double *alpha = &alf;
      const double *beta = &bet;

     // Create a handle for CUBLAS
     hipblasHandle_t handle;
     hipblasCreate(&handle);

     // Do the actual multiplication
     hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

     // Destroy the handle
     hipblasDestroy(handle);

   }
}
