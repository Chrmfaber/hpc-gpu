
#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" { void matmult_gpulib(int m, int n, int k,double *A, double *B, double *C) {

     // Create a handle for CUBLAS
   	 hipblasHandle_t handle;
   	 //cublasCreate(&handle);

      const double alf = 1.0;
      const double bet = 0.0;
      const double *alpha = &alf;
      const double *beta = &bet;

      int lda = k, ldb = n, ldc = n;

     // Do the multiplication
     hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, B, ldb, A, lda, beta, C, ldc);

     // Destroy the handle
     hipblasDestroy(handle);

   }
}
