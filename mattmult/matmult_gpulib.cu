
#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" { void matmult_gpulib(double *A, double *B, double *C, int m, int k, int n) {

     // Create a handle for CUBLAS
   	 hipblasHandle_t handle;
   	 hipblasCreate(&handle);

      const double alf = 1;
      const double bet = 0;

      const double *alpha = &alf;
      const double *beta = &bet;

      int lda = m, ldb = k, ldc = m;

     // Do the multiplication
     hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, A, lda, B, ldb, beta, C, ldc);

     // Destroy the handle
     hipblasDestroy(handle);

   }
}
