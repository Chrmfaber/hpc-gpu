#include <hipblas.h>
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <cstdlib>
#include <cstdio>

<<<<<<< HEAD
extern "C" { void matmult_gpulib(const double *A, const double *B, double *C, const int m, const int k, const int n) {
=======
extern "C" { void matmult_gpulib(double *h_A, double *h_B, double *h_C, int m, int k, int n) {
>>>>>>> 636a2332b58e75c3948451cd22db6f85ccd42592
      int lda=m,ldb=k,ldc=m;
      double alfa = 1.0;
      double beta = 0.0;

     	int size_A = k*n*sizeof(double);
     	int size_B = m*k*sizeof(double);
     	int size_C = m*n*sizeof(double);

      /* Allocate device memory */
      double *d_A, *d_B, *d_C;
      hipMalloc((void **)&d_A, size_A);
      hipMalloc((void **)&d_B, size_B);
      hipMalloc((void **)&d_C, size_C);

      /* Copy data to device */
      hipMemcpy(d_A,h_A,size_A, hipMemcpyHostToDevice);
      hipMemcpy(d_B,h_B,size_B, hipMemcpyHostToDevice);
      hipMemcpy(d_C,h_C,size_C, hipMemcpyHostToDevice);

     // Create a handle for CUBLAS
     hipblasHandle_t handle;
     hipblasCreate(&handle);

     // Do the actual multiplication
     hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);

     hipMemcpy(h_C,d_C,size_C, hipMemcpyDeviceToHost);

     // Destroy the handle
     hipblasDestroy(handle);

   }
<<<<<<< HEAD
}
=======
 }
>>>>>>> 636a2332b58e75c3948451cd22db6f85ccd42592
