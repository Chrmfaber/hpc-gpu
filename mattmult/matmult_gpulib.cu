
#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" { void matmult_gpulib(double *A, double *B, double *C, int m, int n, int k) {

     // Create a handle for CUBLAS
   	 hipblasHandle_t handle;
   	 hipblasCreate(&handle);

      const double alf = 1.0;
      const double bet = 0.0;
      const double *alpha = &alf;
      const double *beta = &bet;

      int lda = m, ldb = k, ldc = m;

     // Do the multiplication
     hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

     // Destroy the handle
     hipblasDestroy(handle);

   }
}
