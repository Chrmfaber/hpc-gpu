
#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" { void matmult_gpulib(int m, int n, int k,double *A, double *B, double *C) {

  int lda = k, ldb = n, ldc = n;

  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  const double alf = 1.0;
  const double bet = 0.0;
  const double *alpha = &alf;
  const double *beta = &bet;

  /* Allocate device memory */
  int size_A = k*m*sizeof(double);
  int size_B = n*k*sizeof(double);
  int size_C = n*m*sizeof(double);

  double *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_B, size_B);
  hipMalloc((void **)&d_C, size_C);

  /* Copy data to device */
  hipMemcpy(d_A,A,size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B,B,size_B, hipMemcpyHostToDevice);
  hipMemcpy(d_C,C,size_C, hipMemcpyHostToDevice);

  // Do the multiplication
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, d_B, ldb, d_A, lda, beta, d_C, ldc);

  // Destroy the handle
  hipblasDestroy(handle);

  // Copy result back to host
  hipMemcpy(C,d_C,size_C, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
}

