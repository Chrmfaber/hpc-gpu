#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <stdio.h>

#define BLOCK_SIZE 16

// A: stride = k
// B: stride = n
// C: stride = n

// Get a matrix element
__device__ float GetElement(float *A, int row, int col, int stride) {
  // printf("Index at = %d \n", row * stride + col);
  return A[row * stride + col];
}

// Set a matrix element
__device__ void SetElement(float *A, int row, int col, int stride,
                           float value) {
  A[row * stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ void GetSubMatrix(float *A, float **Asub, int row, int col,
                             int stride) {
  *Asub = &A[stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
}

__global__ void d_gpu6(int m, int n, int k, float *A, float *B, float *C) {

  int i, e;
  float sum;

  // Block row and column
  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  // Each thread block computes one sub-matrix Csub of C
  float *Csub = &C[n * BLOCK_SIZE * blockRow + BLOCK_SIZE * blockCol];
  // GetSubMatrix(C, &Csub, blockRow, blockCol, n);

  // Each thread computes one element of Csub
  // by accumulating results into Cvalue
  float Cvalue = 0;

  // Thread row and column within Csub
  int row = threadIdx.y;
  int col = threadIdx.x;

  for (i = 0; i < (k / BLOCK_SIZE); ++i) {

    float *Asub = &A[k * BLOCK_SIZE * blockRow + BLOCK_SIZE + i];
    float *Bsub = &B[n * BLOCK_SIZE * i + BLOCK_SIZE + blockCol];
    // Get sub-matrix Asub of A

    // GetSubMatrix(A, &Asub, blockRow, i, k);

    // Get sub-matrix Bsub of B
    // GetSubMatrix(B, &Bsub, i, blockCol, n);

    // Shared memory used to store Asub and Bsub respectively
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load Asub and Bsub from device memory to shared memory
    // Each thread loads one element of each sub-matrix

    As[row][col] = Asub[row * k + col];
    Bs[row][col] = Bsub[row * n + col];

    // Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();
    // Multiply Asub and Bsub together
    for (e = 0; e < BLOCK_SIZE; ++e) {
      Cvalue += As[row][e] * Bs[e][col];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write Csub to device memory
  // Each thread writes one element
  Csub[row * n + col] = Cvalue;
}

extern "C" {
__host__ void matmult_gpu6(int m, int n, int k, float *h_A, float *h_B,
                           float *h_C) {
  float *d_A, *d_B, *d_C;

  hipSetDevice(1);

  int size_A = m * k * sizeof(float);
  int size_B = k * n * sizeof(float);
  int size_C = m * n * sizeof(float);

  hipHostRegister(h_A, size_A);
  hipHostRegister(h_B, size_B);
  hipHostRegister(h_C, size_C);

  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_B, size_B);
  hipMalloc((void **)&d_C, size_C);

  hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(m / dimBlock.x, n / dimBlock.y);

  float time_start_gpu5 = omp_get_wtime();

  d_gpu5<<<dimGrid, dimBlock>>>(m, n, k, d_A, d_B, d_C);

  hipDeviceSynchronize();

  float gpu5_time = omp_get_wtime() - time_start_gpu5;

  hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
}
