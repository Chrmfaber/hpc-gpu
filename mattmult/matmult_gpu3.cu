#include "hip/hip_runtime.h"
__global__ void gpu3_row(int m, int n, int k_max, double *A, double *B, double *C) {

    int i, j, k;
    double sum1, sum2;

    i = 2*(blockIdx.x * blockDim.x + threadIdx.x);
    j = blockIdx.y * blockDim.y + threadIdx.y;

    if(!(i >= m || j >= n)){
         sum1 = 0.0;
         sum2 = 0.0;
         for(k = 0; k < k_max; k++){
            sum1 += A[i*k_max+k] * B[k*n+j];
            sum2 += A[(i+1)*k_max+k] * B[k*n+j];
         }
         C[i*n+j] = sum1;
         C[(i+1)*n+j] = sum2;
      }
}
__global__ void gpu3_column(int m, int n, int k_max, double *A, double *B, double *C) {

    int i, j, k;
    double sum1, sum2;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = 2*(blockIdx.y * blockDim.y + threadIdx.y);

    if(!(i >= m || j >= n)){
         sum1 = 0.0;
         sum2 = 0.0;
         for(k = 0; k < k_max; k++){
            sum1 += A[i*k_max+k] * B[k*n+j];
            sum2 += A[i*k_max+k] * B[k*n+(j+1)];
         }
         C[i*n+j] = sum1;
         C[i*n+(j+1)] = sum2;
      }else if(!(i = m || j = n-1)){
           sum1 = 0.0;
           //sum2 = 0.0;
           for(k = 0; k < k_max; k++){
              sum1 += A[i*k_max+k] * B[k*n+j];
              //sum2 += A[i*k_max+k] * B[k*n+(j+1)];
           }
           C[i*n+j] = sum1;
           //C[i*n+(j+1)] = sum2;
        }
}

extern "C" { __host__ void matmult_gpu3(int m, int n, int k, double *h_A, double *h_B, double *h_C){
   double *d_A, *d_B, *d_C;

   int devices;
   hipGetDeviceCount(&devices);

   int A_elems = m*k;
   int B_elems = k*n;
   int C_elems = m*n;

   int size_A = A_elems*sizeof(double);
   int size_B = B_elems*sizeof(double);
   int size_C = C_elems*sizeof(double);

   hipMalloc((void**)&d_A, size_A);
   hipMalloc((void**)&d_B, size_B);
   hipMalloc((void**)&d_C, size_C);

   hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

   int blockx = 16;
   int blocky = 16;
   dim3 dimBlock(blockx,blocky,1);
   int gridx = (m/blockx)+1;
   int gridy = ((n/blocky)+1)/2 + 1;
   //int gridx = ((m/blockx)+1)/2 + 1;
   //int gridy = (n/blocky)+1;
   dim3 dimGrid(gridx,gridy,1);

   gpu3_column<<<dimGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);
   //gpu3_row<<<dimGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);

   hipDeviceSynchronize();
   hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
   }
}
