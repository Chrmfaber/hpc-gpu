#include "hip/hip_runtime.h"
#define ELEMS 32

__global__ void gpu4_row(int m, int n, int k_max, double *A, double *B, double *C) {

    int i, j, k, l;
    double sum[ELEMS];

    i = ELEMS*(blockIdx.x * blockDim.x + threadIdx.x);
    j = blockIdx.y * blockDim.y + threadIdx.y;

    if(!(i+ELEMS >= m || j >= n)){
         for(l = 0; l < ELEMS; l++){
            sum[l] = 0.0;
         }
         for(k = 0; k < k_max; k++){
            for(l = 0; l < ELEMS; l++){
               sum[l] += A[(i+l)*k_max+k] * B[k*n+j];
            }
            //sum1 += A[i*k_max+k] * B[k*n+j];
            //sum2 += A[(i+1)*k_max+k] * B[k*n+j];
         }
         //C[i*n+j] = sum1;
         //C[(i+1)*n+j] = sum2;
         for(l = 0; l < ELEMS; l++){
            C[(i+l)*n+j] = sum[l];
         }
   }else if(!(i >= m || j >= n)){ //THIS IS CLEANUP IF ELEMS DOES NOT DIVIDE INTO DIMENSIONS
      for(l = 0; i+l < m && l < ELEMS; l++){
         sum[l] = 0.0;
      }
      for(k = 0; k < k_max; k++){
         for(l = 0; i+l < m && l < ELEMS; l++){
            sum[l] += A[(i+l)*k_max+k] * B[k*n+j];
         }
      }
      for(l = 0; i+l < m && l < ELEMS; l++){
         C[(i+l)*n+j] = sum[l];
      }
   }

}
__global__ void gpu4_column(int m, int n, int k_max, double *A, double *B, double *C) {

    int i, j, k, l;
    double sum[ELEMS];

    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = ELEMS*(blockIdx.y * blockDim.y + threadIdx.y);

    if(!(i >= m || j+ELEMS >= n)){
      for(l = 0; l < ELEMS; l++){
          sum[l] = 0.0;
      }
      for(k = 0; k < k_max; k++){
         for(l = 0; l < ELEMS; l++){
            sum[l] += A[i*k_max+k] * B[k*n+(j+l)];
         }
            //sum1 += A[i*k_max+k] * B[k*n+j];
            //sum2 += A[i*k_max+k] * B[k*n+(j+1)];
      }
      for(l = 0; l < ELEMS; l++){
         C[i*n+(j+l)] = sum[l];
      }
         //C[i*n+j] = sum1;
         //C[i*n+(j+1)] = sum2;
      }else if(!(i >= m || j >= n)){
         for(l = 0; j+l < n && l < ELEMS; l++){
             sum[l] = 0.0;
         }
         for(k = 0; k < k_max; k++){
            for(l = 0; j+l < n && l < ELEMS; l++){
               sum[l] += A[i*k_max+k] * B[k*n+(j+l)];
            }
               //sum1 += A[i*k_max+k] * B[k*n+j];
               //sum2 += A[i*k_max+k] * B[k*n+(j+1)];
         }
         for(l = 0; j+l < n && l < ELEMS; l++){
            C[i*n+(j+l)] = sum[l];
         }
      }
}

extern "C" { __host__ void matmult_gpu4(int m, int n, int k, double *h_A, double *h_B, double *h_C){
   double *d_A, *d_B, *d_C;

   int devices;
   hipGetDeviceCount(&devices);

   int A_elems = m*k;
   int B_elems = k*n;
   int C_elems = m*n;

   int size_A = A_elems*sizeof(double);
   int size_B = B_elems*sizeof(double);
   int size_C = C_elems*sizeof(double);

   hipMalloc((void**)&d_A, size_A);
   hipMalloc((void**)&d_B, size_B);
   hipMalloc((void**)&d_C, size_C);

   hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

   int blockx = 16;
   int blocky = 16;
   dim3 dimBlock(blockx,blocky,1);

   //Grid for column algorithm
   int gridx = (m/blockx)+1;
   int gridy = ((n/blocky)+1)/ELEMS + 1;

   //Grid for row algorithm
   //int gridx = ((m/blockx)+1)/ELEMS + 1;
   //int gridy = (n/blocky)+1;
   dim3 dimGrid(gridx,gridy,1);

   double time_start_gpu4 = omp_get_wtime();
   //gpu4_column<<<dimGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);
   gpu4_row<<<dimGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);

   hipDeviceSynchronize();

   double gpu4_time = omp_get_wtime()-time_start_gpu4;

   hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
   /*
   printf("\n");
   for(i = 0;i < m; i++){
      for(j = 0;j < k; j++){
         printf("%f ", h_A[i*k+j]);
      }
      printf("\n");
   }
   printf("\n");
   for(i = 0;i < k; i++){
      for(j = 0;j < n; j++){
         printf("%f ", h_B[i*k+j]);
      }
      printf("\n");
   }
   printf("\n");
   for(i = 0;i < m; i++){
      for(j = 0;j < n; j++){
         printf("%f ", h_C[i*k+j]);
      }
      printf("\n");
   }
*/

  printf("GPUTime = %f\n", gpu4_time);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
   }
}
