extern "C" {

#include <hip/hip_runtime.h>
#include <cblas.h>
#include <omp.h>
#include <stdio.h>

void matmult_lib(int m, int n, int k, double *A, double *B, double *C) {

    double alpha = 1.0;
    double beta = 0.0;

    int lda = k, ldb = n, ldc = n;


    /* Ensure that all elements of C is 0 */
  	for (int i=0; i<m ; i++){
  		for (int j=0; j<n ; j++){
  			C[i*n + j] = 0;
  		}

  	}
    double time_start_lib = omp_get_wtime();
  cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
  double lib_time = omp_get_wtime()-time_start_lib;

  printf("CPUTime = %f\n", lib_time);
}
}
