extern "C" {

#include <hip/hip_runtime.h>
#include <cblas.h>

void matmult_lib(int m, int n, int k, double *A, double *B, double *C) {

    double alpha = 1.0;
    double beta = 0.0;

    int lda = k, ldb = n, ldc = n;


    /* Ensure that all elements of C is 0 */
  	for (int i=0; i<m ; i++){
  		for (int j=0; j<n ; j++){
  			C[i*n + j] = 0;
  		}

  	}
  cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
}
