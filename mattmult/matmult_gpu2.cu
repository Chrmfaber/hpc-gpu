
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>
__global__ void gpu2(int m, int n, int k_max, double *A, double *B, double *C) {

    int i, j, k;
    double sum;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;

    if(!(i >= m || j >= n)){
         sum = 0.0;
         for(k = 0; k < k_max; k++){
            //sum += A[i*k_max+k] * B[k*n+j];
            sum += A[i*k_max+k] * B[k*n+j];
         }
         C[i*n+j] = sum;
         //printf("%d %d %f\n", i,j,sum);
      }
}


extern "C" {__host__ void matmult_gpu2(int m, int n, int k, double *h_A, double *h_B, double *h_C){
   double *d_A, *d_B, *d_C;

   int devices;
   hipGetDeviceCount(&devices);

   int A_elems = m*k;
   int B_elems = k*n;
   int C_elems = m*n;

   int size_A = A_elems*sizeof(double);
   int size_B = B_elems*sizeof(double);
   int size_C = C_elems*sizeof(double);

   hipMalloc((void**)&d_A, size_A);
   hipMalloc((void**)&d_B, size_B);
   hipMalloc((void**)&d_C, size_C);

   hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

   int blockx = 16;
   int blocky = 16;
   dim3 dimBlock(blockx,blocky,1);
   int gridx = (n/blockx)+1;
   int gridy = (m/blocky)+1;
   dim3 dimGrid(gridx,gridy,1);

   double time_start_gpu2 = omp_get_wtime();

   gpu2<<<dimGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);

   hipDeviceSynchronize();

   double gpu2_time = omp_get_wtime()-time_start_gpu2;

   hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
   int i,j;
   /*
   printf("\n");
   for(i = 0;i < m; i++){
      for(j = 0;j < k; j++){
         printf("%f ", h_A[i*k+j]);
      }
      printf("\n");
   }
   printf("\n");
   for(i = 0;i < k; i++){
      for(j = 0;j < n; j++){
         printf("%f ", h_B[i*k+j]);
      }
      printf("\n");
   }
   printf("\n");
   for(i = 0;i < m; i++){
      for(j = 0;j < n; j++){
         printf("%f ", h_C[i*k+j]);
      }
      printf("\n");
   }
   */

   printf("GPUTime = %f\n", gpu2_time);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
   }
}
