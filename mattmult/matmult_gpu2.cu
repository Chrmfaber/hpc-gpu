
#include <hip/hip_runtime.h>
__global__ void gpu2(int m, int n, int p, double *A, double *B, double *C) {

    int i, j, k;
    double sum;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;

    if(!(i >= m || j >= n)){
         sum = 0.0;
         for(k = 0; k < p; k++){
            sum += A[i*p+k] * B[k*m+j];
         }
         C[i*n+j] = sum;
      }
}


extern "C" {__host__ void matmult_gpu2(int m, int n, int k, double *h_A, double *h_B, double *h_C){
   double *d_A, *d_B, *d_C;

   int devices;
   hipGetDeviceCount(&devices);

   int A_elems = n*k;
   int B_elems = k*m;
   int C_elems = n*m;

   int size_A = n*k*sizeof(double);
   int size_B = k*m*sizeof(double);
   int size_C = n*m*sizeof(double);

   hipMalloc((void**)&d_A, size_A);
   hipMalloc((void**)&d_B, size_B);
   hipMalloc((void**)&d_C, size_C);

   hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

   int blockx = 16;
   int blocky = 16;
   dim3 dimBlock(blockx,blocky,1);
   int gridx = (n/blockx)+1;
   int gridy = (m/blocky)+1;
   dim3 dimGrid(gridx,gridy,1);

   gpu2<<<dimGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);

   hipDeviceSynchronize();
   hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
   }
}
