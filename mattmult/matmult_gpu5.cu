#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime_api.h>

#define BLOCK_SIZE 16


// A: stride = k
// B: stride = n
// C: stride = n

// Get a matrix element
__device__ double GetElement(double *A, int row, int col, int stride) {
  //printf("Index at = %d \n", row * stride + col);
  return A[row * stride + col];
}

// Set a matrix element
__device__ void SetElement(double *A, int row, int col, int stride,
                           double value) {
  A[row * stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ void GetSubMatrix(double *A, double **Asub, int row, int col, int stride) {
   *Asub = &A[stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
}

__global__ void d_gpu5(int m, int n, int k, double *A, double *B, double *C) {

  int i, e;
  double sum;

  // Block row and column
  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  // Each thread block computes one sub-matrix Csub of C
  double *Csub;
  GetSubMatrix(C, &Csub, blockRow, blockCol, n);
  // Each thread computes one element of Csub
  // by accumulating results into Cvalue
  float Cvalue = 0;


  // Thread row and column within Csub
  int row = threadIdx.y;
  int col = threadIdx.x;

  for (i = 0; i < (k / BLOCK_SIZE); ++i) {

    double *Asub, *Bsub;
    // Get sub-matrix Asub of A
    GetSubMatrix(A, &Asub, blockRow, i, k);

    // Get sub-matrix Bsub of B
    GetSubMatrix(B, &Bsub, i, blockCol, n);

    // Shared memory used to store Asub and Bsub respectively
    __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load Asub and Bsub from device memory to shared memory
    // Each thread loads one element of each sub-matrix

    As[row][col] = GetElement(Asub, row, col, k);
    Bs[row][col] = GetElement(Bsub, row, col, n);

    // Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();
    // Multiply Asub and Bsub together
    for (e = 0; e < BLOCK_SIZE; ++e) {
      Cvalue += As[row][e] * Bs[e][col];

    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();


  }

  // Write Csub to device memory
  // Each thread writes one element
  SetElement(Csub, row, col, n, Cvalue);

}

extern "C" {
__host__ void matmult_gpu5(int m, int n, int k, double *h_A, double *h_B,
                           double *h_C) {
  double *d_A, *d_B, *d_C;

  hipSetDevice(1);

  int size_A = m * k * sizeof(double);
  int size_B = k * n * sizeof(double);
  int size_C = m * n * sizeof(double);

  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_B, size_B);
  hipMalloc((void **)&d_C, size_C);

  hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(m / dimBlock.x,  n / dimBlock.y);


  double time_start_gpu5 = omp_get_wtime();

  d_gpu5<<<dimGrid, dimBlock>>>(m, n, k, d_A, d_B, d_C);

  hipDeviceSynchronize();

  double gpu5_time = omp_get_wtime()-time_start_gpu5;

  hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

  int i, j;

  printf("GPUTime = %f\n", gpu5_time);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
}
