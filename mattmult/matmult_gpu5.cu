#include "hip/hip_runtime.h"
#define BLOCK_SIZE 16

// A: stride = k
// B: stride = n
// C: stride = n

// Get a matrix element
__device__ float GetElement(double *A, int row, int col, int stride) {
  return A[row * stride + col];
}

// Set a matrix element
__device__ void SetElement(double *A, int row, int col, int stride,
                           float value) {
  A[row * stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ float GetSubMatrix(double *A, int row, int col, int stride) {
  return A[stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
}

__global__ void d_gpu5(int m, int n, int p, double *A, double *B, double *C) {

  int i, e;
  double sum;

  // Block row and column
  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  // Each thread block computes one sub-matrix Csub of C
  double *Csub = GetSubMatrix(C, blockRow, blockCol, n);
  // Each thread computes one element of Csub
  // by accumulating results into Cvalue
  float Cvalue = 0;

  // Thread row and column within Csub
  int row = threadIdx.y;
  int col = threadIdx.x;

  for (i = 0; i < (k / BLOCK_SIZE); ++i) {

    // Get sub-matrix Asub of A
    Matrix Asub = GetSubMatrix(A, blockRow, i, k);

    // Get sub-matrix Bsub of B
    Matrix Bsub = GetSubMatrix(B, i, blockCol, n);

    // Shared memory used to store Asub and Bsub respectively
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load Asub and Bsub from device memory to shared memory
    // Each thread loads one element of each sub-matrix
    As[row][col] = GetElement(Asub, row, col, k);
    Bs[row][col] = GetElement(Bsub, row, col, n);

    // Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();
    // Multiply Asub and Bsub together
    for (int e = 0; e < BLOCK_SIZE; ++e)
      Cvalue += As[row][e] * Bs[e][col];

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write Csub to device memory
  // Each thread writes one element
  SetElement(Csub, row, col, n, Cvalue);
}

extern "C" {
__host__ void matmult_gpu5(int m, int n, int k, double *h_A, double *h_B,
                           double *h_C) {
  double *d_A, *d_B, *d_C;

  int devices;
  hipGetDeviceCount(&devices);

  int A_elems = n * k;
  int B_elems = k * m;
  int C_elems = n * m;

  int size_A = n * k * sizeof(double);
  int size_B = k * m * sizeof(double);
  int size_C = n * m * sizeof(double);

  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_B, size_B);
  hipMalloc((void **)&d_C, size_C);

  hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(k / dimBlock.x, k / dimBlock.y);

  d_gpu5<<<dimGrid, dimBlock>>>(m, n, k, d_A, d_B, d_C);

  hipDeviceSynchronize();
  hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
}
